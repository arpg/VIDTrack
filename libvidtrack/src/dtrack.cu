#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <limits>
//#include <float.h>
//#include <string>

#include <vidtrack/dtrack.cuh>


///////////////////////////////////////////////////////////////////////////
///  Device Functions
///////////////////////////////////////////////////////////////////////////
__global__ void dEstimate()
{
//  int px = blockDim.x*blockIdx.x + threadIdx.x;
//  int py = blockDim.y*blockIdx.y + threadIdx.y;

}



///////////////////////////////////////////////////////////////////////////
///  Host Functions
///////////////////////////////////////////////////////////////////////////
cuDTrack::cuDTrack(unsigned int max_height, unsigned int max_width)
{
  hipMalloc((void**)&d_ref_image_, max_width*max_height*sizeof(unsigned char));
  hipMalloc((void**)&d_ref_depth_, max_width*max_height*sizeof(float));
  hipMalloc((void**)&d_live_image_, max_width*max_height*sizeof(unsigned char));

  // Storage for Least Squares System
  // 21 for upper diagonal of LHS, 6 for RHS, 1 for squared_error, 1 for num_obs.
  hipMalloc((void**)&d_lss_, max_width*max_height*sizeof(float)*29);

#if 0
  hipMalloc((void**)&lss_.jacobian, max_width*max_height*sizeof(float)*6);
  hipMalloc((void**)&lss_.error, max_width*max_height*sizeof(float));
  hipMalloc((void**)&lss_.weight, max_width*max_height*sizeof(float));
  hipMalloc((void**)&lss_.obs, max_width*max_height*sizeof(bool));
#endif
}

///////////////////////////////////////////////////////////////////////////
cuDTrack::~cuDTrack()
{
  if (d_ref_image_ != NULL) {
    hipFree(d_ref_image_);
  }
  if (d_ref_depth_ != NULL) {
    hipFree(d_ref_depth_);
  }
  if (d_live_image_ != NULL) {
    hipFree(d_live_image_);
  }
  if (d_lss_ != NULL) {
    hipFree(d_lss_);
  }
}

///////////////////////////////////////////////////////////////////////////
void cuDTrack::_LaunchEstimate(unsigned int image_height, unsigned int image_width)
{
  dim3 gridSize, blockSize;
  gridSize.x = _GCD(image_width, 32);
  gridSize.y = _GCD(image_height, 32);
  blockSize.x = image_width / gridSize.x;
  blockSize.y = image_height / gridSize.y;

  dEstimate<<<gridSize, blockSize>>>();

  _CheckErrors("Estimate");
}

///////////////////////////////////////////////////////////////////////////
int cuDTrack::_GCD(int a, int b)
{
  if (a % b == 0) {
    return b; //(a >= b) ? b : a;
  }
  return _GCD(a, a % b);
}

///////////////////////////////////////////////////////////////////////////
void cuDTrack::_CheckErrors(const char* label)
{
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA Error [%s]: %s\n", label, hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

///////////////////////////////////////////////////////////////////////////
unsigned int cuDTrack::_CheckMemory()
{
  hipError_t err;

  size_t avail;
  size_t total;

  err = hipMemGetInfo(&avail, &total);
  if (err != hipSuccess) {
    err = hipDeviceReset();
    if (err != hipSuccess) {
      fprintf(stderr, "CheckMemoryCUDA: Unable to reset device: %s \n", hipGetErrorString(err));
    } else {
      err = hipMemGetInfo(&avail, &total);
    }
  }

  if (err == hipSuccess) {
    size_t used = total - avail;
    const unsigned bytes_per_mb = 1024*1000;
    fprintf(stdout, "- Checking CUDA Memory: Total = %lu,  Available = %lu, Used = %lu\n",
            total/bytes_per_mb, avail/bytes_per_mb, used/bytes_per_mb);
    return avail/bytes_per_mb;
  } else {
    fprintf(stderr, "CheckMemoryCUDA: There is an irrecoverable error: %s\n", hipGetErrorString(err));
    fprintf(stderr, "CheckMemoryCUDA: There is an irrecoverable error: %s\n", hipGetErrorString(err));
  }
  return 0;
}

